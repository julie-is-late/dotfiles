
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
  int nDevices;
  hipGetDeviceCount(&nDevices);

  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Cuda Dev: %d , Dev name: '%s' , Arch: sm_%d%d \n", i, prop.name, prop.major, prop.minor);
  }
}

